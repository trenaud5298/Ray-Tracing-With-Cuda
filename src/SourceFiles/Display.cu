#include "hip/hip_runtime.h"
#include "../HeaderFiles/Display.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <math.h>
#include <stdio.h>
#include <fstream>
#include <chrono>
#include <thread>
#include "../../lib/SDL/include/SDL.h"
#include "../../lib/SDL/include/SDL_ttf.h"

#include "../HeaderFiles/Camera.h"
#include "../HeaderFiles/Button.h"


#define MAIN_MENU 0
#define RUNNING 1
#define PAUSED 2


__device__ float3 getVector(int x,int y,int imageWidth,int imageHeight,float* CameraData, float3* rayOrigin, hiprandState* randomGenerator) {
    float heightToWidthRatio = static_cast<float>(imageHeight)/imageWidth;
    float3 tempRayVector = make_float3(CameraData[10]*(static_cast<float>(2*x-imageWidth)/imageWidth),heightToWidthRatio*(CameraData[10]*static_cast<float>(imageHeight-2*y)/imageHeight),static_cast<float>(CameraData[8]));
    float offsetX = CameraData[9]*hiprand_normal(randomGenerator);
    float offsetY = CameraData[9]*hiprand_normal(randomGenerator);
    tempRayVector.x -= offsetX;
    tempRayVector.y -= offsetY;
    (*rayOrigin).x += offsetX;
    (*rayOrigin).y += offsetY;
    float inverseLength = rsqrtf(tempRayVector.x*tempRayVector.x+tempRayVector.y*tempRayVector.y+tempRayVector.z*tempRayVector.z);
    tempRayVector.x *= inverseLength;
    tempRayVector.y *= inverseLength;
    tempRayVector.z *= inverseLength;

    float verticalZ = tempRayVector.y*CameraData[7] + tempRayVector.z*CameraData[5];
    return make_float3(tempRayVector.x*CameraData[4]+verticalZ*CameraData[6],tempRayVector.y*CameraData[5]-tempRayVector.z*CameraData[7],verticalZ*CameraData[4]-tempRayVector.x*CameraData[6]);
}

__device__ float3 getSkyBoxColor(float3 rayVector, float angleOfSun) {
    /* FUTURE IDEA:
     * Use an Image(s) of a SkyBox and In Python
     * Convert all of the RGB Values for each
     * pixels to floats between 0 and 1 and then
     * store that data in a file to be read in
     * for this program and then store that data in
     * an array and be prepared to convert from
     * a vector to a pixel from the skybox
    */
    float3 skyColor;
    float3 sunColor = make_float3(1.0f,1.0f,1.0f);
    if(angleOfSun > 0) {
        float angleBetweenRayAndSun = acos(rayVector.x*cosf(angleOfSun) + rayVector.y*sinf(angleOfSun));
        if(angleBetweenRayAndSun < 0.05) {
            skyColor.x = sunColor.x;
            skyColor.y = sunColor.y;
            skyColor.z = sunColor.z;
            return skyColor;
        }
        float t = 0.5f*(1.0f+(sinf(acos(rsqrtf(rayVector.x*rayVector.x+rayVector.z*rayVector.z)*(rayVector.x*rayVector.x+rayVector.z*rayVector.z)))));
        float3 topSkyColor = make_float3(64.0f/255.0f,156.0f/255.0f,255.0f/255.0f);
        float3 bottomSkyColor = make_float3(1.0f,1.0f,1.0f);
        skyColor.x = topSkyColor.x*t + bottomSkyColor.x*(1-t);
        skyColor.y = topSkyColor.y*t + bottomSkyColor.y*(1-t);
        skyColor.z = topSkyColor.z*t + bottomSkyColor.z*(1-t);
        return skyColor;

    } else {
        skyColor.x = 0;
        skyColor.y = 0;
        skyColor.z = 0;
        return skyColor;
    }
}

__device__ float computeCrossProductLength(float3 vec1, float3 vec2) {
    return sqrtf(
        powf((vec1.y*vec2.z-vec2.y*vec1.z),2)+
        powf((vec1.z*vec2.x-vec2.z*vec1.x),2)+
        powf((vec1.x*vec2.y-vec2.x*vec1.y),2)
    );
}

__device__ float3 getRandomBounceVector(float3 normalVector, hiprandState* randomGenerator) {
    float3 randomBounceVector = make_float3(1.0f,0,0);
    float x;
    float y;
    float z;
    float squaredLength;
    for(int i = 0; i < 100; i ++) {
        x = hiprand_normal(randomGenerator);
        y = hiprand_normal(randomGenerator);
        z = hiprand_normal(randomGenerator);
        squaredLength = sqrtf(x*x+y*y+z*z); 
        if(squaredLength < 1) {
            randomBounceVector.x = x/squaredLength;
            randomBounceVector.y = y/squaredLength;
            randomBounceVector.z = z/squaredLength;
            break;
        }
    }
    if((normalVector.x*randomBounceVector.x + normalVector.y*randomBounceVector.y + normalVector.z*randomBounceVector.z) < 0) {
        randomBounceVector.x *= -1;
        randomBounceVector.y *= -1;
        randomBounceVector.z *= -1;

    }

    return randomBounceVector;
}

__device__ float3 getCorrectBounceVector(float3 normalVector, float3 rayVector, float intersectionObjectSmoothness,hiprandState* randomGenerator) {
    float3 diffuseBounceRay = make_float3(1.0f,0,0);
    float x;
    float y;
    float z;
    float squaredLength;
    for(int i = 0; i < 100; i ++) {
        x = hiprand_normal(randomGenerator);
        y = hiprand_normal(randomGenerator);
        z = hiprand_normal(randomGenerator);
        squaredLength = sqrtf(x*x+y*y+z*z); 
        if(squaredLength < 1) {
            x += normalVector.x;
            y += normalVector.y;
            z += normalVector.z;
            squaredLength = rsqrtf(x*x+y*y+z*z);
            diffuseBounceRay.x = x*squaredLength;
            diffuseBounceRay.y = y*squaredLength;
            diffuseBounceRay.z = z*squaredLength;
            break;
        }
    }
    
    float3 specularBounceRay;
    float doubleDotProduct = 2.0 * (rayVector.x*normalVector.x + rayVector.y*normalVector.y + rayVector.z*normalVector.z);
    specularBounceRay = make_float3 (
        rayVector.x - (doubleDotProduct*normalVector.x),
        rayVector.y - (doubleDotProduct*normalVector.y),
        rayVector.z - (doubleDotProduct*normalVector.z)
    );
    float inverseLength = 1/sqrtf(specularBounceRay.x*specularBounceRay.x + specularBounceRay.y*specularBounceRay.y + specularBounceRay.z*specularBounceRay.z);
    specularBounceRay.x *= inverseLength;
    specularBounceRay.y *= inverseLength;
    specularBounceRay.z *= inverseLength;



    float3 actualBounceRay = make_float3(
        specularBounceRay.x*intersectionObjectSmoothness + diffuseBounceRay.x*(1-intersectionObjectSmoothness),
        specularBounceRay.y*intersectionObjectSmoothness + diffuseBounceRay.y*(1-intersectionObjectSmoothness),
        specularBounceRay.z*intersectionObjectSmoothness + diffuseBounceRay.z*(1-intersectionObjectSmoothness)
    );

    return actualBounceRay;
}

__device__ float3 getCorrectBounceVector2(float3 normalVector, float3 rayVector, float intersectionObjectSmoothness,hiprandState* randomGenerator) {
    float3 diffuseBounceRay = make_float3(1.0f,0,0);
    float x;
    float y;
    float z;
    float squaredLength;
    float randomVal = hiprand_uniform(randomGenerator);
    float3 actualBounceRay;
    if(randomVal < intersectionObjectSmoothness) {
        x/*Double Dot Product*/ = 2.0 * (rayVector.x*normalVector.x + rayVector.y*normalVector.y + rayVector.z*normalVector.z);
        actualBounceRay = make_float3(
            rayVector.x - (x*normalVector.x),
            rayVector.y - (x*normalVector.y),
            rayVector.z - (x*normalVector.z)
        );
    } else {
        for(int i = 0; i < 100; i ++) {
            x = hiprand_normal(randomGenerator);
            y = hiprand_normal(randomGenerator);
            z = hiprand_normal(randomGenerator);
            squaredLength = sqrtf(x*x+y*y+z*z); 
            if(squaredLength < 1) {
                x += normalVector.x;
                y += normalVector.y;
                z += normalVector.z;
                squaredLength = rsqrtf(x*x+y*y+z*z);
                actualBounceRay.x = x*squaredLength;
                actualBounceRay.y = y*squaredLength;
                actualBounceRay.z = z*squaredLength;
                break;
            }
        }
    }
    return actualBounceRay;
}

__device__ float* handleRayIntersection(float3 rayOrigin, float3 rayVector, float**worldObjectData, int numWorldObjects, bool* intersected, float* shortestIntersectionDistance) {
    float* intersectionObject;
    float deltaX,deltaY,deltaZ,B,C,discriminant,distance;
    float3 intersectionPoint;
    for(int i = 0; i < numWorldObjects; i++) {
        switch(static_cast<int>(worldObjectData[i][0])) {
            case 0:
                deltaX = rayOrigin.x - worldObjectData[i][6];
                deltaY = rayOrigin.y - worldObjectData[i][7];
                deltaZ = rayOrigin.z - worldObjectData[i][8];
                B = 2*deltaX*rayVector.x + 2*deltaY*rayVector.y+2*deltaZ*rayVector.z;
                C = deltaX*deltaX + deltaY*deltaY + deltaZ*deltaZ - (worldObjectData[i][9]*worldObjectData[i][9]);
                discriminant = (B*B)-(4*C);
                if(discriminant < 0){break;}
                distance = (-B-sqrtf(discriminant))*(0.5);
                
                if(distance > 0 ){
                    if(!*intersected) {
                        *intersected = true;
                        *shortestIntersectionDistance = distance;
                        intersectionObject = worldObjectData[i];
                    } else if(distance < *shortestIntersectionDistance){
                        *intersected = true;
                        *shortestIntersectionDistance = distance;
                        intersectionObject = worldObjectData[i];
                    }
                }
                break;

            
            case 1:
                distance = 
                (worldObjectData[i][9]-worldObjectData[i][6]*rayOrigin.x-worldObjectData[i][7]*rayOrigin.y-worldObjectData[i][8]*rayOrigin.z)
                /(worldObjectData[i][6]*rayVector.x + worldObjectData[i][7]*rayVector.y + worldObjectData[i][8]*rayVector.z);
                if(distance < 0 || isnan(distance) || distance > *shortestIntersectionDistance) {break;}
                //Determine if point is in triangle
                intersectionPoint.x = rayOrigin.x + rayVector.x*distance;
                intersectionPoint.y = rayOrigin.y + rayVector.y*distance;
                intersectionPoint.z = rayOrigin.z + rayVector.z*distance;
                //AB Inside Test
                if(((intersectionPoint.x-worldObjectData[i][10])*worldObjectData[i][19]  +  
                (intersectionPoint.y-worldObjectData[i][11])*worldObjectData[i][20]  +  
                (intersectionPoint.z-worldObjectData[i][12])*worldObjectData[i][21]) < 0) {break;}

                //BC Inside Test
                if(((intersectionPoint.x-worldObjectData[i][13])*worldObjectData[i][22]  +  
                (intersectionPoint.y-worldObjectData[i][14])*worldObjectData[i][23]  +  
                (intersectionPoint.z-worldObjectData[i][15])*worldObjectData[i][24]) < 0) {break;}

                //AC Inside Test
                if(((intersectionPoint.x-worldObjectData[i][16])*worldObjectData[i][25]  +  
                (intersectionPoint.y-worldObjectData[i][17])*worldObjectData[i][26]  +  
                (intersectionPoint.z-worldObjectData[i][18])*worldObjectData[i][27]) < 0) {break;}

                
                *shortestIntersectionDistance = distance;
                intersectionObject = worldObjectData[i];
                *intersected = true;
                
                break;
        }
    }
    return intersectionObject;
}





__device__ float3 getNormalVector(float* intersectionObject, float3 intersectionPoint, float3 rayVector) {
    float3 normalVector;
    switch(static_cast<int>(intersectionObject[0])) {
        case 0:
            normalVector.x = intersectionPoint.x-intersectionObject[6];
            normalVector.y = intersectionPoint.y-intersectionObject[7];
            normalVector.z = intersectionPoint.z-intersectionObject[8];
            break;

        case 1:
            normalVector.x = intersectionObject[6];
            normalVector.y = intersectionObject[7];
            normalVector.z = intersectionObject[8];
            break;
    }
    if((normalVector.x*rayVector.x + normalVector.y*rayVector.y + normalVector.z*rayVector.z)>0) {
        normalVector.x*=-1;
        normalVector.y*=-1;
        normalVector.z*=-1;
    }
    //NORMALIZE VECTOR:
    float inverseLength = rsqrtf(normalVector.x*normalVector.x + normalVector.y*normalVector.y + normalVector.z*normalVector.z);
    normalVector.x *= inverseLength;
    normalVector.y *= inverseLength;
    normalVector.z *= inverseLength;
    return normalVector;

}


__global__ void DiffuseRender(unsigned int seed, int frame,int maxRayBounces, int imageWidth, int imageHeight,int numWorldObjects, float* cameraData, float* imageDataArray, float** worldObjectData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        hiprandState randomGenerator;
        hiprand_init(seed+frame*79231,x+y*imageWidth*blockDim.z,0,&randomGenerator);
        //Rotate Vector:
        float3 rayOrigin = make_float3(cameraData[0],cameraData[1],cameraData[2]);
        float3 rayVector = getVector(x,y,imageWidth,imageHeight,cameraData,&rayOrigin,&randomGenerator);
        float3 tempColor;
        float3 color = make_float3(1.0,1.0,1.0);
        float3 lightColor = make_float3(0.0f,0.0f,0.0f);
        float colorScalar;
        float3 lightRecieved = {0.0f,0.0f,0.0f};
        float3 normalVector;
        bool intersected;
        float shortestIntersectionDistance;
        float* intersectionObject;
        for(int i =0; i < maxRayBounces; i ++) {
            intersected = false;
            shortestIntersectionDistance = 999999999999;
            intersectionObject = handleRayIntersection(rayOrigin,rayVector,worldObjectData,numWorldObjects,&intersected,&shortestIntersectionDistance);
            if(intersected){
                tempColor.x = intersectionObject[1];
                tempColor.y = intersectionObject[2];
                tempColor.z = intersectionObject[3];
                if(intersectionObject[5] > 0.5) {
                    lightRecieved.x += tempColor.x * color.x;
                    lightRecieved.y += tempColor.y * color.y;
                    lightRecieved.z += tempColor.z * color.z;
                }
                colorScalar = 1.0f/fmaxf(1.0f,fmaxf(tempColor.x,fmaxf(tempColor.y,tempColor.z)));
                color.x *= (tempColor.x*colorScalar);
                color.y *= (tempColor.y*colorScalar);
                color.z *= (tempColor.z*colorScalar);

               
                rayOrigin.x += rayVector.x*shortestIntersectionDistance;
                rayOrigin.y += rayVector.y*shortestIntersectionDistance;
                rayOrigin.z += rayVector.z*shortestIntersectionDistance;
                normalVector = getNormalVector(intersectionObject,rayOrigin,rayVector);
                rayOrigin.x+=normalVector.x*0.000001;
                rayOrigin.y+=normalVector.y*0.000001;
                rayOrigin.z+=normalVector.z*0.000001;
                rayVector = getCorrectBounceVector(normalVector,rayVector,intersectionObject[4],&randomGenerator);
            } else{
                // lightColor = getSkyBoxColor(rayVector,0.3);
                break;
            }
        }
        
        imageDataArray[pixelIndex] = imageDataArray[pixelIndex] + lightRecieved.x; // Red
        imageDataArray[pixelIndex + 1] = imageDataArray[pixelIndex+1] + lightRecieved.y; // Green
        imageDataArray[pixelIndex + 2] = imageDataArray[pixelIndex+2] + lightRecieved.z;
    }
};


__global__ void updateImage(int frame, int imageWidth, int imageHeight,float* imageDataArray,unsigned char* imageData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        float3 pixelColor = make_float3(
            static_cast<float>(imageDataArray[pixelIndex]/static_cast<float>(frame)),
            static_cast<float>(imageDataArray[pixelIndex+1]/static_cast<float>(frame)),
            static_cast<float>(imageDataArray[pixelIndex+2]/static_cast<float>(frame))
        );

        float maxColorVal = fmaxf(pixelColor.x,fmaxf(pixelColor.y,pixelColor.z));
        float colorScale = 1;
        if(maxColorVal > 1) {
            colorScale = 1/maxColorVal;
        }
        
        imageData[pixelIndex] = static_cast<unsigned char>(255.99f*colorScale*pixelColor.x); // Red
        imageData[pixelIndex + 1] = static_cast<unsigned char>(255.99f*colorScale*pixelColor.y); // Green
        imageData[pixelIndex + 2] = static_cast<unsigned char>(255.99f*colorScale*pixelColor.z); // Blue
    }
};

__global__ void updateImageHDR(int frame, int imageWidth, int imageHeight,float* imageDataArray,unsigned char* imageData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        float3 pixelColor = make_float3(
            static_cast<float>(imageDataArray[pixelIndex]/static_cast<float>(frame)),
            static_cast<float>(imageDataArray[pixelIndex+1]/static_cast<float>(frame)),
            static_cast<float>(imageDataArray[pixelIndex+2]/static_cast<float>(frame))
        );

        
        //Tone Map
        float luminance = 0.2126 * pixelColor.x + 0.7152 * pixelColor.y + 0.0722 * pixelColor.z;
        float mapped_luminance = luminance / (1.0f + luminance);
        float scaleFactor;
        if(luminance > 0) {
            scaleFactor = mapped_luminance / luminance;
            pixelColor.x *= scaleFactor;
            pixelColor.y *= scaleFactor;
            pixelColor.z *= scaleFactor;
        }
       
        // //Limits Max Color to 1
        // float maxColorVal = fmaxf(pixelColor.x,fmaxf(pixelColor.y,pixelColor.z));
        // float colorScale = 1;
        // if(maxColorVal > 1) {
        //     colorScale = 1/maxColorVal;
        // }
        // pixelColor.x*= colorScale;
        // pixelColor.y*= colorScale;
        // pixelColor.z*= colorScale;

        
        
        //Apply Gamma Correction
        float inverseGamma = 1.0f/2.2f;
        pixelColor.x = powf(pixelColor.x,inverseGamma);
        pixelColor.y = powf(pixelColor.y,inverseGamma);
        pixelColor.z = powf(pixelColor.z,inverseGamma);
        
        

        imageData[pixelIndex] = static_cast<unsigned char>(255.99f*fminf(1.0,pixelColor.x)); // Red
        imageData[pixelIndex + 1] = static_cast<unsigned char>(255.99f*fminf(1.0,pixelColor.y)); // Green
        imageData[pixelIndex + 2] = static_cast<unsigned char>(255.99f*fminf(1.0,pixelColor.z)); // Blue
        
    }
};


__global__ void newRender(unsigned int seed, int frame, int maxRayBounces, int imageWidth, int imageHeight,int numWorldObjects, float* cameraData, float* rawImageDataArray, unsigned char* actualImageDataArray, float** worldObjectData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        hiprandState randomGenerator;
        hiprand_init(seed+frame*79231,x+y*imageWidth*blockDim.z,0,&randomGenerator);
        //Rotate Vector:
        float3 rayOrigin = make_float3(cameraData[0],cameraData[1],cameraData[2]);
        float3 rayVector = getVector(x,y,imageWidth,imageHeight,cameraData,&rayOrigin,&randomGenerator);
        float3 tempColor;
        float3 color = make_float3(1.0,1.0,1.0);
        float3 lightColor = make_float3(0.0f,0.0f,0.0f);
        float colorScalar;
        float3 lightRecieved = {0.0f,0.0f,0.0f};
        float3 normalVector;
        bool intersected;
        float shortestIntersectionDistance;
        float* intersectionObject;
        for(int i =0; i < maxRayBounces; i ++) {
            intersected = false;
            shortestIntersectionDistance = 999999999999;
            intersectionObject = handleRayIntersection(rayOrigin,rayVector,worldObjectData,numWorldObjects,&intersected,&shortestIntersectionDistance);
            if(intersected){
                tempColor.x = intersectionObject[1];
                tempColor.y = intersectionObject[2];
                tempColor.z = intersectionObject[3];
                if(intersectionObject[5] > 0.5) {
                    lightRecieved.x += tempColor.x * color.x;
                    lightRecieved.y += tempColor.y * color.y;
                    lightRecieved.z += tempColor.z * color.z;
                }
                colorScalar = 1.0f/fmaxf(1.0f,fmaxf(tempColor.x,fmaxf(tempColor.y,tempColor.z)));
                color.x *= (tempColor.x*colorScalar);
                color.y *= (tempColor.y*colorScalar);
                color.z *= (tempColor.z*colorScalar);

               
                rayOrigin.x += rayVector.x*shortestIntersectionDistance;
                rayOrigin.y += rayVector.y*shortestIntersectionDistance;
                rayOrigin.z += rayVector.z*shortestIntersectionDistance;
                normalVector = getNormalVector(intersectionObject,rayOrigin,rayVector);
                rayOrigin.x+=normalVector.x*0.000001;
                rayOrigin.y+=normalVector.y*0.000001;
                rayOrigin.z+=normalVector.z*0.000001;
                rayVector = getCorrectBounceVector2(normalVector,rayVector,intersectionObject[4],&randomGenerator);
            } else{
                // lightColor = getSkyBoxColor(rayVector,0.3);
                break;
            }
        }


        float proportionalConstant = 1.0/static_cast<float>(frame);

        float3 updatedPixelColor = make_float3(
            rawImageDataArray[pixelIndex]   * (1-proportionalConstant) + lightRecieved.x * proportionalConstant,
            rawImageDataArray[pixelIndex+1] * (1-proportionalConstant) + lightRecieved.y * proportionalConstant,
            rawImageDataArray[pixelIndex+2] * (1-proportionalConstant) + lightRecieved.z * proportionalConstant
        );

        rawImageDataArray[pixelIndex]   = updatedPixelColor.x;
        rawImageDataArray[pixelIndex+1] = updatedPixelColor.y;
        rawImageDataArray[pixelIndex+2] = updatedPixelColor.z;

        // float maxColorVal = fmaxf(updatedPixelColor.x,fmaxf(updatedPixelColor.y,updatedPixelColor.z));
        // float colorScale = 1;
        // if(maxColorVal > 1) {
        //     colorScale = 1/maxColorVal;
        // }
        
        // actualImageDataArray[pixelIndex] = static_cast<unsigned char>(255.99f*colorScale*updatedPixelColor.x); // Red
        // actualImageDataArray[pixelIndex + 1] = static_cast<unsigned char>(255.99f*colorScale*updatedPixelColor.y); // Green
        // actualImageDataArray[pixelIndex + 2] = static_cast<unsigned char>(255.99f*colorScale*updatedPixelColor.z); // Blue


        //HDR TONE MAP
        // //Tone Map Colors
        float luminance = 0.2126 * updatedPixelColor.x + 0.7152 * updatedPixelColor.y + 0.0722 * updatedPixelColor.z;
        float mapped_luminance = luminance / (1.0f + luminance);
        float scaleFactor;
        if(luminance > 0) {
            scaleFactor = mapped_luminance / luminance;
            updatedPixelColor.x *= scaleFactor;
            updatedPixelColor.y *= scaleFactor;
            updatedPixelColor.z *= scaleFactor;
        }

        //Apply Gamma Correction
        float inverseGamma = 1.0f/2.2f;
        updatedPixelColor.x = powf(updatedPixelColor.x,inverseGamma);
        updatedPixelColor.y = powf(updatedPixelColor.y,inverseGamma);
        updatedPixelColor.z = powf(updatedPixelColor.z,inverseGamma);

        
        actualImageDataArray[pixelIndex] = static_cast<unsigned char>(255.99f*fminf(1.0,updatedPixelColor.x)); // Red
        actualImageDataArray[pixelIndex + 1] = static_cast<unsigned char>(255.99f*fminf(1.0,updatedPixelColor.y)); // Green
        actualImageDataArray[pixelIndex + 2] = static_cast<unsigned char>(255.99f*fminf(1.0,updatedPixelColor.z)); // Blue
        
    }
};







//Need to Redesign how function works (consider using structs)
__global__ void DiffuseRender2(unsigned int seed, int frame, int maxRayBounces, int imageWidth, int imageHeight,int numWorldObjects, float* cameraData, float* rawImageDataArray, float** worldObjectData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        hiprandState randomGenerator;
        hiprand_init(seed+frame*79231,x+y*imageWidth*blockDim.z,0,&randomGenerator);
        //Rotate Vector:
        float3 initialRayOrigin = make_float3(cameraData[0],cameraData[1],cameraData[2]);
        float3 initialRayVector = getVector(x,y,imageWidth,imageHeight,cameraData,&initialRayOrigin,&randomGenerator);
        float3 initialColor = {0.0f,0.0f,0.0f};
        float3 tempColor = {0.0f,0.0f,0.0f};
        float3 lightRecieved = {0.0f,0.0f,0.0f};
        float3 initialNormalVector;
        bool intersected;
        float shortestIntersectionDistance;
        float* intersectionObject;
        float colorScalar;
        //Determine the initial intersection Point:
        intersected = false;
        shortestIntersectionDistance = 999999999999;
        intersectionObject = handleRayIntersection(initialRayOrigin,initialRayVector,worldObjectData,numWorldObjects,&intersected,&shortestIntersectionDistance);
        if(intersected) {
        
            //Handle Starting Color/Light
            tempColor.x = intersectionObject[1];
            tempColor.y = intersectionObject[2];
            tempColor.z = intersectionObject[3];
            if(intersectionObject[5] > 0.5) {
                lightRecieved.x += tempColor.x;
                lightRecieved.y += tempColor.y;
                lightRecieved.z += tempColor.z;
            }
            colorScalar = 1.0f/fmaxf(1.0f,fmaxf(tempColor.x,fmaxf(tempColor.y,tempColor.z)));
            initialColor.x = (tempColor.x*colorScalar);
            initialColor.y = (tempColor.y*colorScalar);
            initialColor.z = (tempColor.z*colorScalar);
            //Shifts Origin To First Intersection Point
            initialRayOrigin = {
                initialRayOrigin.x + initialRayVector.x*shortestIntersectionDistance,
                initialRayOrigin.y + initialRayVector.y*shortestIntersectionDistance,
                initialRayOrigin.z + initialRayVector.z*shortestIntersectionDistance
            };
            initialNormalVector = getNormalVector(intersectionObject,initialRayOrigin,initialRayVector);
            initialRayOrigin.x += initialNormalVector.x*1e-6;
            initialRayOrigin.y += initialNormalVector.y*1e-6;
            initialRayOrigin.z += initialNormalVector.z*1e-6;

            float smoothness = intersectionObject[4];
            //Handles 10 Unique Bounces From Initial Point
            float3 rayVector;
            float3 rayOrigin;
            float3 normalVector;
            float3 bounceColor;
            for(int i = 0; i < 10; i ++) {
                rayVector = getCorrectBounceVector2(initialNormalVector,initialRayVector,smoothness,&randomGenerator);
                rayOrigin = initialRayOrigin;
                bounceColor = initialColor;
                for(int j = 0; j < maxRayBounces-1; j++) {
                    intersected = false;
                    shortestIntersectionDistance = 999999999999;
                    intersectionObject = handleRayIntersection(rayOrigin,rayVector,worldObjectData,numWorldObjects,&intersected,&shortestIntersectionDistance);
                    if(intersected){
                        tempColor.x = intersectionObject[1];
                        tempColor.y = intersectionObject[2];
                        tempColor.z = intersectionObject[3];
                        if(intersectionObject[5] > 0.5) {
                            lightRecieved.x += 0.1f * tempColor.x * bounceColor.x;
                            lightRecieved.y += 0.1f * tempColor.y * bounceColor.y;
                            lightRecieved.z += 0.1f * tempColor.z * bounceColor.z;
                        }
                        colorScalar = 1.0f/fmaxf(1.0f,fmaxf(tempColor.x,fmaxf(tempColor.y,tempColor.z)));
                        bounceColor.x *= (tempColor.x*colorScalar);
                        bounceColor.y *= (tempColor.y*colorScalar);
                        bounceColor.z *= (tempColor.z*colorScalar);

                    
                        rayOrigin.x += rayVector.x*shortestIntersectionDistance;
                        rayOrigin.y += rayVector.y*shortestIntersectionDistance;
                        rayOrigin.z += rayVector.z*shortestIntersectionDistance;
                        normalVector = getNormalVector(intersectionObject,rayOrigin,rayVector);
                        rayOrigin.x+=normalVector.x*0.000001;
                        rayOrigin.y+=normalVector.y*0.000001;
                        rayOrigin.z+=normalVector.z*0.000001;
                        rayVector = getCorrectBounceVector2(normalVector,rayVector,intersectionObject[4],&randomGenerator);
                    } else{
                        // lightColor = getSkyBoxColor(rayVector,0.3);
                        break;
                    }

                }
            }
        }

        rawImageDataArray[pixelIndex]    =  rawImageDataArray[pixelIndex]    +  lightRecieved.x; // Red
        rawImageDataArray[pixelIndex+1]  =  rawImageDataArray[pixelIndex+1]  +  lightRecieved.y; // Green
        rawImageDataArray[pixelIndex+2]  =  rawImageDataArray[pixelIndex+2]  +  lightRecieved.z;
        
    }
};







Display::Display(int displayWidth,int displayHeight,Camera* viewCamera): displayWidth(displayWidth),displayHeight(displayHeight),viewCamera(viewCamera),displayState(MAIN_MENU),currentFrame(0) {
    this->imageData = new unsigned char[this->displayWidth * this->displayHeight * 3];
    std::cout<<"Display Created"<<std::endl;

    //Initializes SDL VIDEO
    if (SDL_Init(SDL_INIT_VIDEO) != 0 || TTF_Init() == -1) {
        std::cout<<"MAJOR ERROR: FAILURE TO LOAD SDL"<<std::endl;
    }

    //Make Display Window
    this->displayWindow = SDL_CreateWindow("RayTracing", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, this->displayWidth, this->displayHeight, SDL_WINDOW_RESIZABLE);
    if (this->displayWindow==NULL) {
        std::cout<<"MAJOR ERROR: FAILURE TO CREATE SDL WINDOW"<<std::endl;
    }
    this->resetCursor();
    
    //Initialize and Create SDL Renderer to display image
    this->imageRenderer = SDL_CreateRenderer(this->displayWindow, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    SDL_SetRenderDrawBlendMode(this->imageRenderer, SDL_BLENDMODE_BLEND);
    
    this->imageTexture = SDL_CreateTexture(this->imageRenderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, this->displayWidth, this->displayHeight);

    this->allocateGPUMemory();
    this->recieveGpuTroubleShootData = new float[30];
    std::random_device rd;
    this->randomSeedGenerator.seed(rd());
    this->seedDistribution = std::uniform_real_distribution<float>(0,1);
};



Display::~Display() {
    std::cout<<"Display Deconstructing"<<std::endl;
    if(this->rawGpuImageData)
        hipFree(this->rawGpuImageData);
    if(this->gpuImage)
        hipFree(this->gpuImage);
    if(this->gpuCameraData)
        hipFree(this->gpuCameraData);
    if(this->displayWindow)
        SDL_DestroyWindow(this->displayWindow);
    if(this->imageTexture)
        SDL_DestroyTexture(this->imageTexture);
    
}




void Display::updateDisplayParameters(Settings* settings) {
    this->displayWidth = settings->getGraphicsSettingsValue("ScreenResolutionX");
    this->displayHeight = settings->getGraphicsSettingsValue("ScreenResolutionY");

}



#pragma region GPU MEMORY
void Display::allocateWorldData(World* world) {
    float** worldDataAsArray = world->getWorldObjectDataAsArray();
    this->numOfWorldObjects = world->numOfObjects;
    hipMalloc(&this->gpuWorldObjectData, this->numOfWorldObjects * sizeof(float*));
    for(size_t i = 0; i < this->numOfWorldObjects; i++) {
        float* objectData;
        hipMalloc(&objectData, 30 * sizeof(float));
        hipMemcpy(objectData, worldDataAsArray[i],30*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(&this->gpuWorldObjectData[i],&objectData,sizeof(float*),hipMemcpyHostToDevice);
    }
}



void Display::allocateGPUMemory() {
    hipMalloc(&this->gpuTroubleShootData, 30*sizeof(float));
    hipMalloc(&this->gpuCameraData, 30 * sizeof(float));
    hipMalloc(&this->rawGpuImageData, this->displayWidth * this->displayHeight * 3 * sizeof(float));
    hipMalloc(&this->gpuImage, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char));
}

void Display::reAllocateGPUMemory() {
    hipFree(this->gpuImage);
    hipFree(this->rawGpuImageData);
    hipFree(this->gpuCameraData);
    hipMalloc(&this->gpuCameraData, 30 * sizeof(float));
    hipMalloc(&this->rawGpuImageData, this->displayWidth * this->displayHeight * 3 * sizeof(float));
    hipMalloc(&this->gpuImage, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char));
}

void Display::updateGPUData() {
    hipMemcpy(this->gpuCameraData, this->viewCamera->cameraData, 30 * sizeof(float), hipMemcpyHostToDevice);
}

void Display::copyImageData() {
    hipMemcpy(this->gpuImage, this->imageData, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char),hipMemcpyHostToDevice);
}
#pragma endregion




void Display::getPixelInfo(int x, int y) {
    int index = (y * this->displayWidth + x) * 3;
    std::cout << "Pixel (" << x << ", " << y << "): ( " << static_cast<int>(this->imageData[index]) << ", " << static_cast<int>(this->imageData[index+1]) << ", " << static_cast<int>(this->imageData[index+2]) << " )" << std::endl;
}

void Display::resetCursor() {
    SDL_WarpMouseInWindow(this->displayWindow, this->displayWidth / 2, this->displayHeight / 2);
    this->previousMousePosX = this->displayWidth/2;
    this->previousMousePosY = this->displayHeight/2;
    this->mousePosX = this->displayWidth/2;
    this->mousePosY = this->displayHeight/2;
}

/*
NEXT STEP IS TO REORGANIZE EVENTS INTO THE EVENT MANAGER CLASS
NEEDED QUITE BADLY

*/




void Display::renderScene() {
    this->renderImage();
    SDL_RenderClear(this->imageRenderer);
    SDL_UpdateTexture(this->imageTexture, NULL, this->imageData, this->displayWidth*3);
    SDL_RenderCopy(this->imageRenderer, this->imageTexture, NULL, NULL);
    SDL_RenderPresent(this->imageRenderer);
}


void Display::handleDisplaySizeChange(int newDisplayWidth, int newDisplayHeight) {
    this->displayWidth = newDisplayWidth;
    this->displayHeight = newDisplayHeight;
    delete[] this->imageData;
    this->imageData = new unsigned char[this->displayWidth * this->displayHeight * 3];
    SDL_DestroyTexture(this->imageTexture);
    this->imageTexture = SDL_CreateTexture(this->imageRenderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, this->displayWidth, this->displayHeight);
    hipFree(this->gpuImage);
    hipMalloc(&this->gpuImage, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char));

}




void Display::renderImage() {
    dim3 threads(16, 8);
    dim3 blocksPerGrid(ceil(displayWidth / static_cast<float>(threads.x)),
                        ceil(displayHeight / static_cast<float>(threads.y)));

    //Update GPU Data
    if(this->currentFrame == 0){
        this->updateGPUData();
    }
    this->currentFrame += 1;

    int randomGenSeed = static_cast<int>(99999.0*this->seedDistribution(this->randomSeedGenerator));
    

    // Launch the CUDA kernel defined inside the class
    newRender<<<blocksPerGrid, threads>>>(randomGenSeed, this->currentFrame, 10,displayWidth, displayHeight, this->numOfWorldObjects,this->gpuCameraData,this->rawGpuImageData,this->gpuImage, this->gpuWorldObjectData,this->gpuTroubleShootData);

    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
        delete this; // Free allocated memory
        return;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy the processed image back from GPU to CPU
    hipMemcpy(this->imageData, this->gpuImage, displayWidth * displayHeight * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    // hipMemcpy(this->recieveGpuTroubleShootData,this->gpuTroubleShootData,30*sizeof(float),hipMemcpyDeviceToHost);
    // std::cout<<std::endl<<"------------"<<std::endl;
    // for(int i = 0; i < 7; i ++) {
    //     std::cout<<"Value "<<i<<": "<<this->recieveGpuTroubleShootData[i]<<std::endl;
    // }
}


void Display::seriousRenderOfImage(int numOfFrames,int maxRayBounces) {
    dim3 threads(16, 8);
    dim3 blocksPerGrid(ceil(displayWidth / static_cast<float>(threads.x)),
                        ceil(displayHeight / static_cast<float>(threads.y)));

    float* imageDataArray = new float[this->displayWidth*this->displayHeight*3];
    float* gpuImageDataArray;
    hipMalloc(&gpuImageDataArray,this->displayWidth*this->displayHeight*3*sizeof(float));

    for(int frame = 1; frame<=numOfFrames; frame++) {
        std::cout<<"Rendering Frame: "<<frame<<std::endl;
        //Update GPU Data
        this->updateGPUData();
        this->copyImageData();

        int randomGenSeed = static_cast<int>(99999.0*this->seedDistribution(this->randomSeedGenerator));
        
        
        // Launch the CUDA kernel defined inside the class
        DiffuseRender<<<blocksPerGrid, threads>>>(randomGenSeed,frame,maxRayBounces,displayWidth, displayHeight, this->numOfWorldObjects,this->gpuCameraData,gpuImageDataArray, this->gpuWorldObjectData,this->gpuTroubleShootData);
        
        hipError_t kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        updateImage<<<blocksPerGrid, threads>>>(frame,displayWidth, displayHeight,gpuImageDataArray,this->gpuImage,this->gpuTroubleShootData);
        kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy the processed image back from GPU to CPU
        hipMemcpy(this->imageData, this->gpuImage, displayWidth * displayHeight * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        //Updates DISPLAYED Image
        SDL_RenderClear(this->imageRenderer);
        SDL_UpdateTexture(this->imageTexture, NULL, this->imageData, this->displayWidth*3);
        SDL_RenderCopy(this->imageRenderer, this->imageTexture, NULL, NULL);
        SDL_RenderPresent(this->imageRenderer);
    }
}



void Display::customResolutionRender(std::string fileName, int numOfFrames,int maxRayBounces, int resolutionX, int resolutionY) {
    dim3 threads(16, 8);
    dim3 blocksPerGrid(ceil(resolutionX / static_cast<float>(threads.x)),
                        ceil(resolutionY / static_cast<float>(threads.y)));

    float* imageDataArray = new float[resolutionX*resolutionY*3];
    float* gpuImageDataArray;
    hipMalloc(&gpuImageDataArray,resolutionX*resolutionY*3*sizeof(float));

    unsigned char* customResImageData = new unsigned char[resolutionX*resolutionY*3];
    unsigned char* customResGPUImageData;
    hipMalloc(&customResGPUImageData,resolutionX*resolutionY*3*sizeof(unsigned char));

    hipError_t kernelError;
    auto startTime = std::chrono::high_resolution_clock::now();
    for(int frame = 1; frame<=numOfFrames; frame++) {
        std::cout<<"Rendering Frame: "<<frame<<std::endl;
        //Update GPU Data
        this->updateGPUData();
        this->copyImageData();

        int randomGenSeed = static_cast<int>(99999.0*this->seedDistribution(this->randomSeedGenerator));
        
        
        // Launch the CUDA kernel defined inside the class
        DiffuseRender2<<<blocksPerGrid, threads>>>(randomGenSeed,frame,maxRayBounces,resolutionX, resolutionY, this->numOfWorldObjects,this->gpuCameraData,gpuImageDataArray, this->gpuWorldObjectData,this->gpuTroubleShootData);
        
        kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();
    }
    auto endTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = endTime - startTime;
    std::cout<<"Time taken: "<<duration.count()<<" seconds"<<std::endl;

    updateImageHDR<<<blocksPerGrid, threads>>>(numOfFrames,resolutionX, resolutionY,gpuImageDataArray,customResGPUImageData,this->gpuTroubleShootData);
        kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy the processed image back from GPU to CPU
    hipMemcpy(customResImageData, customResGPUImageData, resolutionX*resolutionY * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    std::cout<<"Swapping Red and Blue Values (SDL EXPECATION OF ORDER)"<<std::endl;
    for (int y = 0; y < resolutionY; ++y) {
        for (int x = 0; x < resolutionX; ++x) {
            unsigned char temp = customResImageData[(y * resolutionX + x) * 3];
            customResImageData[(y * resolutionX + x) * 3] = customResImageData[(y * resolutionX + x) * 3 + 2];
            customResImageData[(y * resolutionX + x) * 3 + 2] = temp;
        }
    }
    std::cout<<"saving image..."<<std::endl;
    // Write image data to BMP file
    SDL_Surface* surface = SDL_CreateRGBSurfaceFrom(customResImageData, resolutionX, resolutionY, 24, resolutionX * 3, 0xFF0000, 0x00FF00, 0x0000FF, 0);
    std::string filePath = "./Saved Images/" +fileName + ".bmp";
    std::cout<<"FilePath: "<<filePath<<std::endl;
    SDL_SaveBMP(surface,filePath.c_str());
}


void Display::saveImage(const std::string& fileName) {
    std::cout<<"saving image..."<<std::endl;
    // Write image data to BMP file
    SDL_Surface* surface = SDL_CreateRGBSurface(0, displayWidth, displayHeight, 32, 0, 0, 0, 0);
    SDL_RenderReadPixels(this->imageRenderer,NULL,SDL_PIXELFORMAT_ARGB8888, surface->pixels, surface->pitch);

    std::string filePath = "./Saved Images/" +fileName + ".bmp";
    std::cout<<"FilePath: "<<filePath<<std::endl;
    SDL_SaveBMP(surface,filePath.c_str());
    
}

